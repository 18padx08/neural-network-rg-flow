#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>
#include "utils.cuh"
#include "rbm_helpers.cuh"

using namespace utils;

__global__
void contrastive_divergence(hiprandState *globalState,int *input, double *weights, double *bh, double *bv, bool *mask, double *ph_mean, int *ph_sample, double *nv_means, int *nv_samples, double *nh_means, int *nh_samples, int n_hidden, int n_visible, double lr, int N) {
	//printf("in cuda kernel");
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if (row > n_visible || col > n_hidden) return;

	/* CD-k */
	//mtx.lock();
	//printf("start CD \n");

		hiprandState state = globalState[row*blockDim.y * 32 + col];
	sample_h_given_v(state,input, ph_mean, ph_sample, bh, weights, n_hidden, n_visible);
	__syncthreads();
	//sample_v_given_h(state, ph_sample, nv_means, nv_samples, weights, bv, n_visible, n_hidden);
	for (int step = 0; step<1; step++) {
		if (step == 0) {
			//printf("only do it once \n");
			gibbs_hvh(state,ph_sample, nv_means, nv_samples, nh_means, nh_samples, weights, bv, bv, n_visible, n_hidden);
		}
		
	}
	__syncthreads();
	
	if (mask[row*n_hidden + col]) {
		//printf("yes it was masked \n");
		//printf("%d, ", ph_mean[i]); printf("%d, ", input[j]); printf("%d, ", nh_means[i]); printf("%d, ", nv_samples[j]);
		weights[row*n_hidden + col] += lr * (ph_mean[col] * input[row] - nh_means[col] * nv_samples[row]) / N;
		//bh[col] += lr * (ph_sample[col] - nh_means[col]) / N;
		//bv[row] += lr * (input[row] - nv_samples[row]) / N;
		//printf("|%d|", weights[i*n_visible + j]);
	}
	else {
		//penalty if not masked
		//weights[row*n_hidden + col] += lr * lr * lr* (ph_mean[col] * input[row] - nh_means[col] * nv_samples[row]) / N;
		//bh[col] += lr *lr * (ph_sample[col] - nh_means[col]) / N;
		//bv[row] += lr*lr * (input[row] - nv_samples[row]) / N;
	}
}

__device__
void sample_h_given_v(hiprandState globalState,int *v0_sample, double *mean, int *sample, double *hbias, double *weights, int n_hidden, int n_visible) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if (row > n_visible || col > n_hidden) return;
		double val = propup(v0_sample, weights, hbias[col], n_visible, n_hidden);
		mean[col] = val;
		int state = binomial(1, mean[col], globalState);
		sample[col] = state;
		//printf("%f -> %d\n",val,state);
	
}

__device__
void sample_v_given_h(hiprandState globalState,int *h0_sample, double *mean, int *sample, double *weights, double *vbias, int n_visible, int n_hidden) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if (row > n_visible || col > n_hidden) return;

	mean[row] = propdown(h0_sample, vbias[row],n_hidden,n_visible,weights);
	sample[row] = binomial(1, mean[row], globalState);
	
}

__device__
double propup(int *v, double *w, double bh, int n_visible, int n_hidden) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if (row > n_visible || col > n_hidden) return;

	double pre_sigmoid_activation = 0.0;
	for (int j = 0; j<n_visible; j++) {
		pre_sigmoid_activation +=v[j] * w[j*n_hidden + col];
	}
	//pre_sigmoid_activation += bh;
	return sigmoid(pre_sigmoid_activation);
}

__device__
double propdown(int *h, double bv, int n_hidden, int n_visible, double *W) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if (row > n_visible || col > n_hidden) return;
	double pre_sigmoid_activation = 0.0;
	for (int j = 0; j<n_hidden; j++) {
		pre_sigmoid_activation += W[row*n_hidden + j] * h[j];
	}
	//pre_sigmoid_activation += bv;
	return sigmoid(pre_sigmoid_activation);
}

__device__
void gibbs_hvh(hiprandState globalState,int *h0_sample, double *nv_means, int *nv_samples, double *nh_means, int *nh_samples, double *weights, double *vbias, double *hbias, int n_visible, int n_hidden) {
	sample_v_given_h(globalState,h0_sample, nv_means, nv_samples, weights, vbias, n_visible, n_hidden);
	__syncthreads();
	sample_h_given_v(globalState,nv_samples, nh_means, nh_samples, hbias, weights, n_hidden, n_visible);
}
