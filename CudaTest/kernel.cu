#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "RBM.cuh"
#include "Ising1D.h"
#include <fstream>
#include <iostream>
#include <thread>



__host__
void generateSpins(int **samples);
__host__
void readSpins(int **samples);
__host__
void theIsing(int **samples);

#define NUM_THREADS 4

int main()
{
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}

	srand(time(NULL));
	int **samples = (int **)malloc(1000 * sizeof(int*));

	generateSpins(samples);
	
	
	//why 3x3?
	bool *mask = (bool*)malloc(500 * 1000* sizeof(bool));
	int maskCounter = 0;
	int lastRow = 2;
	int lastCol = 0;
	bool second = false;
	mask[0] = true;
	for (int i = 0; i < 1000; i++) {
		for (int j = 0; j < 500; j++) {
			if (i % 2 == 0)
			{
				
				if (i==lastRow && j == lastCol) {
					mask[i * 500 + j] = true;
					maskCounter++;
					if (!second) {
						lastCol += 1;
						second = true;
					}
				}
				else {
					if (i == 0 && j == 0) {
						mask[0] = true;
					}
					else {
						mask[i * 500 + j] = false;
					}
				}
			}
			else {
				mask[i * 500 + j] = false;
			}
		
		}
		if (i % 2 == 0 && i > 0) {
			second = false;
			lastRow += 2;
		}
	}
	printf("%d values masked\n", maskCounter);

	

	RBM rbm(1000,500,mask);
	rbm.train(samples, 40);
	rbm.printWeights();
	
    return 0;
}

void theIsing(int **samples) {
	for (int i = 0; i < 10; i++) {
		Ising1D ising(1000, 1, 1);
		int counter = 0;
		double mE, tE, M;
		do {
			ising.monteCarloStep();
			counter++;
			mE = ising.getMeanEnergy();
			tE = ising.getTheoreticalMeanEnergy();
			M = ising.getMagnetization();

		} while (!(abs(tE - mE) < 0.1 * abs(tE) && abs(ising.getMagnetization()) < 0.01));

		if (i % 2 == 0) {
			printf("[STEP %d] Mean energy config: %f theoretical: %f delta: %f\n Mean magnetization: %f\n", i, ising.getMeanEnergy(), ising.getTheoreticalMeanEnergy(), ising.getMeanEnergy() - ising.getTheoreticalMeanEnergy(), ising.getMagnetization());
		}
		samples[i] = (int *)malloc(1000 * sizeof(int *));

		std::vector<int> v = ising.getConfiguration();
		std::copy(v.begin(), v.end(), samples[i]);
	}
}
void generateSpins(int **samples) {
	std::thread t1(theIsing, samples);
	std::thread t2(theIsing, samples + 10);
	std::thread t3(theIsing, samples + 20);
	std::thread t4(theIsing, samples + 30);
	
	t1.join();
	t2.join();
	t3.join();
	t4.join();
	

	std::ofstream spinsOut("spins.csv");

	for (int i = 0; i < 40; i++) {
		for (int j = 0; j < 1000; j++) {
			spinsOut << samples[i][j] << ", ";
		}
		spinsOut << std::endl;
	}
	spinsOut.close();
}

void readSpins(int **samples) {
	char yorno = scanf("Generate new samples?[y/N] %c");
	if (yorno == 'y') {
		generateSpins(samples);
		return;
	}
	std::ifstream input("spins.csv");
	//while(line = input.read)
}
