#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include "RBM.cuh"
#include "rbm_helpers.cuh"
#include <iostream>
#include <fstream>
#include <hiprand/hiprand_kernel.h>
#include <ctime>

__global__ void initialise_curand_on_kernels(hiprandState * state, unsigned long long seed);

double uniform(double min, double max) {
	return rand() / (RAND_MAX + 1.0) * (max - min) + min;
}

RBM::RBM(int n_visible, int n_hidden, bool * mask) :mean(0), mask(mask), bh((double *)malloc(n_hidden * sizeof(double))), bv((double*)malloc(n_visible * sizeof(double ))), n_hidden(n_hidden), n_visible(n_visible)
{
	weights = (double *)malloc(n_hidden * n_visible * sizeof(double));
	for (int i = 0; i < n_hidden*n_visible; i++) {
		if(mask[i])
			weights[i] = 0.66;
	}
}

RBM::~RBM()
{
	free(weights);
	free(bh);
	free(bv);
}

__global__ void initialise_curand_on_kernels(hiprandState * state, unsigned long long seed)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	hiprand_init(seed,row*32 + col, 0, &state[row*blockDim.y * 32 + col]);
}

void RBM::train(int ** samples, int n_samples)
{
	double *devWeights, *devBv, *devBh;
	double *ph_mean, double *nv_means, double *nh_means;
	int *nh_samples, *nv_samples, *ph_sample, *nv_sample;
	nv_sample = (int*)malloc(n_visible * sizeof(int));
	bool *devMask;
	//make device copys
	hipError_t cudaStatus;
	cudaStatus=hipMalloc((void **)&devWeights, n_hidden * n_visible * sizeof(double));
	//dummy shared
	cudaStatus = hipMalloc((void **)&ph_mean, n_hidden  * sizeof(double));
	cudaStatus = hipMalloc((void **)&ph_sample, n_hidden  * sizeof(int));
	cudaStatus = hipMalloc((void **)&nv_means,  n_visible * sizeof(double));
	cudaStatus = hipMalloc((void **)&nv_samples, n_visible * sizeof(int));
	cudaStatus = hipMalloc((void **)&nh_means, n_hidden * sizeof(double));
	cudaStatus = hipMalloc((void **)&nh_samples, n_hidden * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda malloc failed!");

	}
	cudaStatus=hipMalloc((void **)&devBh, n_hidden * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda malloc failed!");

	}
	cudaStatus=hipMalloc((void **)&devBv, n_visible * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda malloc failed!");

	}
	cudaStatus=hipMalloc((void **)&devMask, n_hidden * n_visible * sizeof(bool));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda malloc failed!");

	}

	cudaStatus=hipMemcpy(devWeights, weights, n_hidden*n_visible * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda memcpy failed!");
		
	}
	cudaStatus=hipMemcpy(devBv, bv, n_visible * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda memcpy failed!");
		
	}
	cudaStatus=hipMemcpy(devBh, bh, n_hidden * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda memcpy failed!");
		
	}
	cudaStatus=hipMemcpy(devMask, mask, n_hidden*n_visible * sizeof(bool), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda memcpy failed!");
		
	}

	

	int maxNum = std::fmax(n_hidden, n_visible);
	dim3 threads(32, 32);

	dim3 blocks((maxNum + 1) / 32, (maxNum + 1) / 32);
	hiprandState *globalState;
	unsigned int theSize = 3*32 * 32 * blocks.x * blocks.y * sizeof(hiprandState);
	printf("globalState size: %d\n", theSize);
	hipMalloc((void **)&globalState,theSize);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "init hiprand failed failed! %d\n");
	}
	unsigned long long seed = static_cast<unsigned long long>(time(NULL));
	initialise_curand_on_kernels<<<blocks, threads>>> (globalState,seed);

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "init hiprand failed failed! %d\n");
	}
	for (int j = 0; j < 100; j++) {
		for (int i = 0; i < n_samples; i++) {
			int *sample;
			hipMalloc((void**)&sample, n_visible * sizeof(int));
			int test = samples[i][n_visible - 1];
			hipMemcpy(sample, samples[i], n_visible * sizeof(int), hipMemcpyHostToDevice);
			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "cuda memcpy failed!");
				break;
			}

			contrastive_divergence <<<blocks, threads>>> (globalState,sample, devWeights, devBh, devBv, devMask, ph_mean, ph_sample, nv_means, nv_samples, nh_means, nh_samples, n_hidden, n_visible, 0.08);
			cudaStatus = hipDeviceSynchronize();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "contrastive failed failed! %d\n", i);
			}

			
			hipFree(sample);
		}
		printf("epoch (%d) \n", j);
		hipMemcpy(nv_sample, nv_samples, n_visible * sizeof(int), hipMemcpyDeviceToHost);
		double mag = 0.0, ene = 0.0;
		for (int i = 0; i < n_visible; i++) {
			int spin2 = nv_sample[i] == 0 ? -1 : 1;
			mag += spin2;
			if (i == 0) {
				int spin1 = nv_sample[n_visible - 1] == 0 ? -1 : 1;
				
				ene += -1.0 *(spin1 * spin2);
			}
			else {
				int spin1 = nv_sample[i-1] == 0 ? -1 : 1;
				int spin2 = nv_sample[i] == 0 ? -1 : 1;
				ene += -1.0 *(spin1 * spin2);
			}
		}
		mag /= n_visible;
		ene /= n_visible;
		printf("mag: %f, ene: %f\n", mag, ene);
	}
	//finished copy back 
	
	
	hipMemcpy(weights,devWeights,  n_hidden*n_visible * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(bv, devBv,  n_visible * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(bh, devBh,  n_hidden * sizeof(double), hipMemcpyDeviceToHost);


	
	hipFree(devWeights);
	hipFree(devBv);
	hipFree(devBh);
	hipFree(devMask);
}

void RBM::printWeights() {
	std::ofstream out("weights.csv");
	int counter = 0;
	for (int i = 0; i < n_visible; i++) {
		for (int j = 0; j < n_hidden; j++) {
			//printf("%f,", weights[i * n_hidden + j]);
			double theVal = weights[i*n_hidden + j];
			if (!isnan(theVal) && abs(theVal) > 0) {
				mean += theVal;
				counter++;
			}
			out << weights[i*n_hidden + j];
			out << ", ";
		}
		out << "\n";
	}
	out.close();
	mean /= counter;
	printf("theMeanJ: %f\n", mean);
}

